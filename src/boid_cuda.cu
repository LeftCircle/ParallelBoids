#include "hip/hip_runtime.h"
#include "boid_cuda.h" // Include the C interface header

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h> // For basic error printing
#include <vector> // Keep for temporary host storage if needed inside C interface

__device__ float3_simple operator+(const float3_simple& a, const float3_simple& b) {
	return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ float3_simple operator-(const float3_simple& a, const float3_simple& b) {
	return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ float3_simple operator*(const float3_simple& a, const float b) {
	return {a.x * b, a.y * b, a.z * b};
}

__device__ float3_simple operator/(const float3_simple& a, const float b) {
	if (b == 0.0f) return {0.0f, 0.0f, 0.0f};
	return {a.x / b, a.y / b, a.z / b};
}

__device__ float3_simple normalize(const float3_simple& v) {
	float length_sq = v.x * v.x + v.y * v.y + v.z * v.z;
	if (length_sq > 0) { // Avoid sqrt(0) and division by zero
		float length = sqrtf(length_sq);
		return {v.x / length, v.y / length, v.z / length};
	}
	return {0.0f, 0.0f, 0.0f};
}

__device__ float length_squared(const float3_simple& v) {
	return v.x * v.x + v.y * v.y + v.z * v.z;
}

__device__ float length(const float3_simple& v) {
	return sqrtf(length_squared(v));
}


__global__ void calculate_boid_forces_kernel(
	const float3_simple* current_positions, // Input
	const float3_simple* current_velocities, // Input
	float3_simple* new_velocities,          // Output
	int num_boids,
	float delta_time,
	float neighbor_distance_sq,
	float separation_weight,
	float alignment_weight,
	float cohesion_weight,
	float max_speed,
	float min_speed, // <-- Add min_speed parameter
	float max_force
) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_boids) {
		float3_simple pos = current_positions[idx];
		float3_simple vel = current_velocities[idx];

		float3_simple separation_force = {0.0f, 0.0f, 0.0f};
		float3_simple alignment_sum = {0.0f, 0.0f, 0.0f};
		float3_simple cohesion_center = {0.0f, 0.0f, 0.0f}; // Center of mass for cohesion
		int neighbor_count = 0;

		// Loop through other boids
		for (int i = 0; i < num_boids; ++i) {
			if (idx == i) continue;

			float3_simple other_pos = current_positions[i];
			float3_simple other_vel = current_velocities[i];

			// Calculate distance squared
			float3_simple diff = pos - other_pos;
			float dist_sq = length_squared(diff);

			// Check if neighbor (use squared distance)
			if (dist_sq > 0 && dist_sq < neighbor_distance_sq) {
				neighbor_count++;

				// 1. Separation: Steer away from neighbors
				separation_force = separation_force + (diff / dist_sq); 
				// 2. Alignment: Steer towards average neighbor velocity
				alignment_sum = alignment_sum + other_vel;

				// 3. Cohesion: Steer towards average neighbor position (center of mass)
				cohesion_center = cohesion_center + other_pos;
			}
		}

		float3_simple total_force = {0.0f, 0.0f, 0.0f};

		if (neighbor_count > 0) {
			// Average separation force (optional, can be strong)
			separation_force = separation_force / neighbor_count;
			separation_force = separation_force - vel; // Steering force
			separation_force = separation_force * separation_weight;

			alignment_sum = alignment_sum / neighbor_count; // Average velocity
			alignment_sum = alignment_sum - vel; // Steering force
			alignment_sum = alignment_sum * alignment_weight;

			cohesion_center = cohesion_center / neighbor_count; // Center of mass
			float3_simple desired_cohesion = cohesion_center - pos; // Vector towards center
			desired_cohesion = desired_cohesion - vel; // Steering force
			desired_cohesion = desired_cohesion * cohesion_weight;

			// --- Combine Forces ---
			total_force = separation_force + alignment_sum + desired_cohesion;

			// --- Limit Force ---
			float force_mag_sq = length_squared(total_force);
			if (force_mag_sq > max_force * max_force) {
				total_force = normalize(total_force) * max_force;
			}
		}

		// --- Apply Force & Update Velocity ---
		// Acceleration = Force / Mass (assume mass = 1 for simplicity)
		float3_simple acceleration = total_force; // If mass is 1
		vel = vel + acceleration * delta_time;

		// --- Limit Speed (Max and Min) ---
		float speed_sq = length_squared(vel);
		if (speed_sq > max_speed * max_speed) {
			vel = normalize(vel) * max_speed;
		} else if (speed_sq < min_speed * min_speed) {
			// Only apply min speed if the boid is actually moving (speed_sq > epsilon)
			// to avoid giving stationary boids a random direction.
			// A very small epsilon prevents issues with floating point inaccuracies near zero.
			float epsilon_sq = 1e-9f; // Square of a small epsilon
			if (speed_sq > epsilon_sq) {
				vel = normalize(vel) * min_speed;
			} else {
				// If velocity is essentially zero, leave it zero.
				vel = {0.0f, 0.0f, 0.0f};
			}
		}

		// Write the final calculated velocity for this boid
		new_velocities[idx] = vel;
	}
}


// --- Helper function to check CUDA errors (standard C version) ---
static int HandleCudaError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s in %s at line %d\n", hipGetErrorString(err), file, line);
		return 1; // Return non-zero on error
	}
	return 0; // Return 0 on success
}
#define CUDA_CHECK(err) do { if (HandleCudaError(err, __FILE__, __LINE__)) return 1; } while(0) // Return error code from calling function


// --- C-style interface function implementation ---
int calculate_boid_update_cuda_c_interface(
	const float3_simple* host_positions,
	const float3_simple* host_current_velocities,
	float3_simple* host_new_velocities, // Output parameter
	int num_boids,
	float delta_time,
	float neighbor_distance,
	float separation_weight,
	float alignment_weight,
	float cohesion_weight,
	float max_speed,
	float min_speed, // <-- Add min_speed parameter
	float max_force
) {
	if (num_boids == 0) {
		return 0; // No work to do, success
	}

	// --- 1. Host data is already prepared and passed as pointers ---

	// --- 2. Allocate GPU Memory ---
	float3_simple* device_positions = nullptr;
	float3_simple* device_current_velocities = nullptr;
	float3_simple* device_new_velocities = nullptr;
	size_t positions_size = num_boids * sizeof(float3_simple);
	size_t velocities_size = num_boids * sizeof(float3_simple);

	// Use CUDA_CHECK macro for error handling
	CUDA_CHECK(hipMalloc(&device_positions, positions_size));
	CUDA_CHECK(hipMalloc(&device_current_velocities, velocities_size));
	CUDA_CHECK(hipMalloc(&device_new_velocities, velocities_size));

	// --- 3. Copy Data Host -> Device ---
	CUDA_CHECK(hipMemcpy(device_positions, host_positions, positions_size, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(device_current_velocities, host_current_velocities, velocities_size, hipMemcpyHostToDevice));

	// --- 4. Launch Kernel ---
	int threads_per_block = 256;
	int blocks_per_grid = (num_boids + threads_per_block - 1) / threads_per_block;
	float neighbor_distance_sq = neighbor_distance * neighbor_distance;

	calculate_boid_forces_kernel<<<blocks_per_grid, threads_per_block>>>(
		device_positions,
		device_current_velocities,
		device_new_velocities,
		num_boids,
		delta_time, // Already float
		neighbor_distance_sq,
		separation_weight,
		alignment_weight,
		cohesion_weight,
		max_speed,
		min_speed, // <-- Pass min_speed to kernel
		max_force
	);

	// Check for kernel launch errors
	CUDA_CHECK(hipGetLastError());
	// Synchronize device to ensure kernel completion before copying back
	CUDA_CHECK(hipDeviceSynchronize());


	// --- 5. Copy Data Device -> Host ---
	// Copy directly into the output pointer provided by the caller
	CUDA_CHECK(hipMemcpy(host_new_velocities, device_new_velocities, velocities_size, hipMemcpyDeviceToHost));

	// --- 6. Free GPU Memory ---
	hipError_t free_err; // Check errors individually during cleanup
	free_err = hipFree(device_positions);
	if (free_err != hipSuccess) fprintf(stderr, "CUDA Error freeing device_positions: %s\n", hipGetErrorString(free_err));
	free_err = hipFree(device_current_velocities);
	if (free_err != hipSuccess) fprintf(stderr, "CUDA Error freeing device_current_velocities: %s\n", hipGetErrorString(free_err));
	free_err = hipFree(device_new_velocities);
	if (free_err != hipSuccess) fprintf(stderr, "CUDA Error freeing device_new_velocities: %s\n", hipGetErrorString(free_err));

	// Even if freeing fails, we might have successfully computed the result.
	// The primary return value indicates success/failure of the computation itself.
	return 0; // Success
}


// --- Implementation for is_cuda_available_c_interface ---
bool is_cuda_available_c_interface() {
	int device_count = 0;
	hipError_t err = hipGetDeviceCount(&device_count);
	if (err != hipSuccess) {
		// Don't print error here, let the caller decide based on the return value
		// fprintf(stderr, "CUDA Error getting device count: %s\n", hipGetErrorString(err));
		return false;
	}
	if (device_count == 0) {
		return false;
	}
	// Could add more checks here (e.g., compute capability) if needed
	return true;
}