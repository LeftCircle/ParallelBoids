#include "hip/hip_runtime.h"
#include "boid_cuda.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h> 
#include <vector> 

__device__ float3_simple operator+(const float3_simple& a, const float3_simple& b) {
	return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ float3_simple operator-(const float3_simple& a, const float3_simple& b) {
	return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ float3_simple operator*(const float3_simple& a, const float b) {
	return {a.x * b, a.y * b, a.z * b};
}

__device__ float3_simple operator/(const float3_simple& a, const float b) {
	if (b == 0.0f) return {0.0f, 0.0f, 0.0f};
	return {a.x / b, a.y / b, a.z / b};
}

__device__ float3_simple normalize(const float3_simple& v) {
	float length_sq = v.x * v.x + v.y * v.y + v.z * v.z;
	if (length_sq > 0) {
		float length = sqrtf(length_sq);
		return {v.x / length, v.y / length, v.z / length};
	}
	return {0.0f, 0.0f, 0.0f};
}

__device__ float length_squared(const float3_simple& v) {
	return v.x * v.x + v.y * v.y + v.z * v.z;
}

__device__ float length(const float3_simple& v) {
	return sqrtf(length_squared(v));
}


__global__ void calculate_boid_forces_kernel(
	const float3_simple* current_positions, // Input
	const float3_simple* current_velocities, // Input
	float3_simple* new_velocities,          // Output
	int num_boids,
	float delta_time,
	float neighbor_distance_sq,
	float separation_weight,
	float alignment_weight,
	float cohesion_weight,
	float max_speed,
	float min_speed,
	float max_force
) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_boids) {
		float3_simple pos = current_positions[idx];
		float3_simple vel = current_velocities[idx];

		float3_simple separation_force = {0.0f, 0.0f, 0.0f};
		float3_simple alignment_sum = {0.0f, 0.0f, 0.0f};
		float3_simple cohesion_center = {0.0f, 0.0f, 0.0f};
		int neighbor_count = 0;

		// Loop through other boids
		for (int i = 0; i < num_boids; ++i) {
			if (idx == i) continue;

			float3_simple other_pos = current_positions[i];
			float3_simple other_vel = current_velocities[i];

			float3_simple diff = pos - other_pos;
			float dist_sq = length_squared(diff);

			if (dist_sq > 0 && dist_sq < neighbor_distance_sq) {
				neighbor_count++;

				separation_force = separation_force + (diff / dist_sq); 
				alignment_sum = alignment_sum + other_vel;
				cohesion_center = cohesion_center + other_pos;
			}
		}

		float3_simple total_force = {0.0f, 0.0f, 0.0f};

		if (neighbor_count > 0) {
			// Average separation force (optional, can be strong)
			separation_force = separation_force / neighbor_count;
			separation_force = separation_force - vel;
			separation_force = separation_force * separation_weight;

			alignment_sum = alignment_sum / neighbor_count; 
			alignment_sum = alignment_sum - vel;
			alignment_sum = alignment_sum * alignment_weight;

			cohesion_center = cohesion_center / neighbor_count; 
			float3_simple desired_cohesion = cohesion_center - pos; 
			desired_cohesion = desired_cohesion - vel;
			desired_cohesion = desired_cohesion * cohesion_weight;

			total_force = separation_force + alignment_sum + desired_cohesion;

			float force_mag_sq = length_squared(total_force);
			if (force_mag_sq > max_force * max_force) {
				total_force = normalize(total_force) * max_force;
			}
		}

		// --- Apply Force & Update Velocity ---
		float3_simple acceleration = total_force; // If mass is 1
		vel = vel + acceleration * delta_time;

		float speed_sq = length_squared(vel);
		if (speed_sq > max_speed * max_speed) {
			vel = normalize(vel) * max_speed;
		} else if (speed_sq < min_speed * min_speed) {
			vel = normalize(vel) * min_speed;
		}
		new_velocities[idx] = vel;
	}
}


// --- Helper function to check CUDA errors (standard C version) ---
static int HandleCudaError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s in %s at line %d\n", hipGetErrorString(err), file, line);
		return 1;
	}
	return 0;
}
#define CUDA_CHECK(err) do { if (HandleCudaError(err, __FILE__, __LINE__)) return 1; } while(0)


// --- C-style interface function implementation ---
int calculate_boid_update_cuda_c_interface(
	const float3_simple* host_positions,
	const float3_simple* host_current_velocities,
	float3_simple* host_new_velocities, // Output parameter
	int num_boids,
	float delta_time,
	float neighbor_distance,
	float separation_weight,
	float alignment_weight,
	float cohesion_weight,
	float max_speed,
	float min_speed,
	float max_force
) {
	if (num_boids == 0) {
		return 0; // No work to do, success
	}
	// Allocate GPU Memory
	float3_simple* device_positions = nullptr;
	float3_simple* device_current_velocities = nullptr;
	float3_simple* device_new_velocities = nullptr;
	size_t positions_size = num_boids * sizeof(float3_simple);
	size_t velocities_size = num_boids * sizeof(float3_simple);

	// Use CUDA_CHECK macro for error handling
	CUDA_CHECK(hipMalloc(&device_positions, positions_size));
	CUDA_CHECK(hipMalloc(&device_current_velocities, velocities_size));
	CUDA_CHECK(hipMalloc(&device_new_velocities, velocities_size));

	// Copy Data Host
	CUDA_CHECK(hipMemcpy(device_positions, host_positions, positions_size, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(device_current_velocities, host_current_velocities, velocities_size, hipMemcpyHostToDevice));

	// Launch Kernel
	int threads_per_block = 256;
	int blocks_per_grid = (num_boids + threads_per_block - 1) / threads_per_block;
	float neighbor_distance_sq = neighbor_distance * neighbor_distance;

	calculate_boid_forces_kernel<<<blocks_per_grid, threads_per_block>>>(
		device_positions,
		device_current_velocities,
		device_new_velocities,
		num_boids,
		delta_time,
		neighbor_distance_sq,
		separation_weight,
		alignment_weight,
		cohesion_weight,
		max_speed,
		min_speed,
		max_force
	);

	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());

	// Copy directly into the output pointer provided by the caller
	CUDA_CHECK(hipMemcpy(host_new_velocities, device_new_velocities, velocities_size, hipMemcpyDeviceToHost));

	//  Free GPU Memory
	hipError_t free_err; // Check errors individually during cleanup
	free_err = hipFree(device_positions);
	if (free_err != hipSuccess) fprintf(stderr, "CUDA Error freeing device_positions: %s\n", hipGetErrorString(free_err));
	free_err = hipFree(device_current_velocities);
	if (free_err != hipSuccess) fprintf(stderr, "CUDA Error freeing device_current_velocities: %s\n", hipGetErrorString(free_err));
	free_err = hipFree(device_new_velocities);
	if (free_err != hipSuccess) fprintf(stderr, "CUDA Error freeing device_new_velocities: %s\n", hipGetErrorString(free_err));
	return 0;
}


// --- Implementation for is_cuda_available_c_interface ---
bool is_cuda_available_c_interface() {
	int device_count = 0;
	hipError_t err = hipGetDeviceCount(&device_count);
	if (err != hipSuccess) {
		return false;
	}
	if (device_count == 0) {
		return false;
	}
	return true;
}